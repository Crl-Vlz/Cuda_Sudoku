
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <set>

const int board[9][9] = { {5, 3, 0, 0, 7, 0, 0, 0, 0},
                 {6, 0, 0, 1, 9, 5, 0, 0, 0},
                 {0, 9, 8, 0, 0, 0, 0, 6, 0},
                 {8, 0, 0, 0, 6, 0, 0, 0, 3},
                 {4, 0, 0, 8, 0, 3, 0, 0, 1},
                 {7, 0, 0, 0, 2, 0, 0, 0, 6},
                 {0, 6, 0, 0, 0, 0, 2, 8, 0},
                 {0, 0, 0, 4, 1, 9, 0, 0, 5},
                 {0, 0, 0, 0, 8, 0, 0, 7, 9} };

bool host_check_general(int cols, int rows) {
    if (cols != 9) {
        printf("\nBoard does not respect 9x9 format");
        return 0;
    }
    if (rows != 9) {
        printf("\nBoard does not respect 9x9 format");
        return 0;
    }
    return 1;
}

bool host_check_row(int * board, int row) {
    int starter = row * 9 ; // Passes n values to start a new row
    std::set<int> row_values;
    for (int i = 0; i < 9; i++) {
        if (board[i + starter] == 0) continue;
        if (row_values.find(board[i + starter]) != row_values.end()) return 0;
        row_values.insert(board[i + starter]);
    }
    return true;
}

bool host_check_col(int * board, int col) {
    int starter = col;
    std::set<int> col_values;
    for (int i = 0; i < 9; i++) {
        if (board[i * 9 + starter] == 0) continue;
        if (col_values.find(board[i * 9 + starter]) != col_values.end()) return 0;
        col_values.insert(board[i * 9 + starter]);
    }
    return true;
}

bool host_check_subsquare(int* board, int sub) {
    int col = (sub % 3) * 3, row = (sub / 3) * 3;
    std::set<int> square_values;
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            int starter = (row + i) * 9 + col + j;
            if (board[starter] == 0) continue;
            if (square_values.find(board[starter]) != square_values.end()) return 0;
            square_values.insert(board[starter]);
        }
    }
    return 1;
}

bool host_check_subsquare(int* board, int col, int row) {
    int sub = col / 3 + (row / 3) * 3;
    return host_check_subsquare(board, sub);
}

bool host_check_all(int* board) {
    bool format = true;
    for (int i = 0; i < 9; i++) {
        format = format && host_check_row(board, i);
    }
    for (int i = 0; i < 9; i++) {
        format = format && host_check_col(board, i);
    }
    for (int i = 0; i < 9; i++) {
        format = format && host_check_subsquare(board, i);
    }
    return format;
}

bool host_check_validity(int* board, int n, int col, int row) {
    for (int i = 0; i < 9; i++)
        if (board[row * 9 + i] == n || board[i * 9 + col] == n) return false;
    int sub_row_start = (row / 3) * 3;
    int sub_col_start = (col / 3) * 3;
    for (int i = sub_row_start; i < sub_row_start + 3; i++)
        for (int j = sub_col_start; j < sub_col_start + 3; j++)
            if (board[i * 9 + j] == n) return false;
    return true;
}

int host_solve_sudoku(int* board) {
    if (!host_check_all(board)) return -1; // Board is not valid
    for (int row = 0; row < 9; row++) {
        for (int col = 0; col < 9; col++) {
            int index = col + row * 9;
            if (!board[index]) {
                for (int i = 1; i <= 9; i++) {
                    if (host_check_validity(board, i, col, row)) {
                        board[index] = i;
                        if (host_solve_sudoku(board)) return 1;
                        else board[index] = 0;
                    }
                }
                return 0;
            }
        }
    }
    return 1;
}

void print_pointer(int* pointer, int size, int max_inline = -1) {
    if (max_inline <= 0) {
        for (int i = 0; i < size; i++) printf("\n%d", pointer[i]);
    }
    else {
        int flag = size / max_inline;
        int vals = size % max_inline == 0 ? flag : flag + 1;
        for (int i = 0; i < vals; i++) {
            printf("\n");
            for (int j = 0; j < max_inline; j++) {
                if (j + i * max_inline >= size) return;
                printf("%d ", pointer[j + i * max_inline]);
            }
        }
    }
}

int main()
{
    int* host_pointer;
    // 81 because sudoku boards always have 81 values
    host_pointer = (int*)malloc(sizeof(int) * 81);
    for (int i = 0; i < 81; i++) {
        int row = i / 9;
        int col = i % 9;
        host_pointer[i] = board[row][col];
    }
    int board_status = host_solve_sudoku(host_pointer);
    if (board_status == -1) printf("\nThe format of the is not valid");
    else if (board_status == 0) printf("\nNo valid solution was found");
    else printf("\nA solution was found");
    //printf("\nThe format of the board is: %d", host_check_all(host_pointer));
    print_pointer(host_pointer, 81, 9);
}
