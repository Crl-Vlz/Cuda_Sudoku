
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <set>

const int board[9][9] = { {5, 3, 0, 0, 7, 0, 0, 0, 0},
                 {6, 0, 0, 1, 9, 5, 0, 0, 0},
                 {0, 9, 8, 0, 0, 0, 0, 6, 0},
                 {8, 0, 0, 0, 6, 0, 0, 0, 3},
                 {4, 0, 0, 8, 0, 3, 0, 0, 1},
                 {7, 0, 0, 0, 2, 0, 0, 0, 6},
                 {0, 6, 0, 0, 0, 0, 2, 8, 0},
                 {0, 0, 0, 4, 1, 9, 0, 0, 5},
                 {0, 0, 0, 0, 8, 0, 0, 7, 9} };

bool host_check_general(int cols, int rows) {
    if (cols != 9) {
        printf("\nBoard does not respect 9x9 format");
        return 0;
    }
    if (rows != 9) {
        printf("\nBoard does not respect 9x9 format");
        return 0;
    }
    return 1;
}

bool host_check_row(int * board, int row) {
    int starter = row * 9 ; // Passes n values to start a new row
    std::set<int> row_values;
    for (int i = 0; i < 9; i++) {
        if (board[i + starter] == 0) continue;
        if (row_values.find(board[i + starter]) != row_values.end()) return 0;
        row_values.insert(board[i + starter]);
    }
    return true;
}

bool host_check_col(int * board, int col) {
    int starter = col;
    std::set<int> col_values;
    for (int i = 0; i < 9; i++) {
        if (board[i * 9 + starter] == 0) continue;
        if (col_values.find(board[i * 9 + starter]) != col_values.end()) return 0;
        col_values.insert(board[i * 9 + starter]);
    }
    return true;
}

bool host_check_subsquare(int* board, int sub) {
    int col = (sub % 3) * 3, row = (sub / 3) * 3;
    std::set<int> square_values;
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            int starter = (row + i) * 9 + col + j;
            if (board[starter] == 0) continue;
            if (square_values.find(board[starter]) != square_values.end()) return 0;
            square_values.insert(board[starter]);
        }
    }
    return 1;
}

void print_pointer(int* pointer, int size, int max_inline = -1) {
    if (max_inline <= 0) {
        for (int i = 0; i < size; i++) printf("\n%d", pointer[i]);
    }
    else {
        int flag = size / max_inline;
        int vals = size % max_inline == 0 ? flag : flag + 1;
        for (int i = 0; i < vals; i++) {
            printf("\n");
            for (int j = 0; j < max_inline; j++) {
                if (j + i * max_inline >= size) return;
                printf("%d ", pointer[j + i * max_inline]);
            }
        }
    }
}

int main()
{
    int* host_pointer;
    // 81 because sudoku boards always have 81 values
    host_pointer = (int*)malloc(sizeof(int) * 81);
    for (int i = 0; i < 81; i++) {
        int row = i / 9;
        int col = i % 9;
        host_pointer[i] = board[row][col];
    }
    bool format = true;
    for (int i = 0; i < 9; i++) {
        format = format && host_check_row(host_pointer, i);
    }
    for (int i = 0; i < 9; i++) {
        format = format && host_check_col(host_pointer, i);
    }
    for (int i = 0; i < 9; i++) {
        format = format && host_check_subsquare(host_pointer, i);
    }
    printf("\nThe format of the board is: %d", format);
    print_pointer(host_pointer, 81, 9);
}
