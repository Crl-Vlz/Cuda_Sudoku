#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <set>

const int board[9][9] = { {5, 3, 0, 0, 7, 0, 0, 0, 0},
                 {6, 0, 0, 1, 9, 5, 0, 0, 0},
                 {0, 9, 8, 0, 0, 0, 0, 6, 0},
                 {8, 0, 0, 0, 6, 0, 0, 0, 3},
                 {4, 0, 0, 8, 0, 3, 0, 0, 1},
                 {7, 0, 0, 0, 2, 0, 0, 0, 6},
                 {0, 6, 0, 0, 0, 0, 2, 8, 0},
                 {0, 0, 0, 4, 1, 9, 0, 0, 5},
                 {0, 0, 0, 0, 8, 0, 0, 7, 9} };

bool host_check_general(int cols, int rows) {
    if (cols != 9) {
        printf("\nBoard does not respect 9x9 format");
        return 0;
    }
    if (rows != 9) {
        printf("\nBoard does not respect 9x9 format");
        return 0;
    }
    return 1;
}

bool host_check_row(int * board, int row) {
    int starter = row * 9 ; // Passes n values to start a new row
    std::set<int> row_values;
    for (int i = 0; i < 9; i++) {
        if (board[i + starter] == 0) continue;
        if (row_values.find(board[i + starter]) != row_values.end()) return 0;
        row_values.insert(board[i + starter]);
    }
    return true;
}

bool host_check_col(int * board, int col) {
    int starter = col;
    std::set<int> col_values;
    for (int i = 0; i < 9; i++) {
        if (board[i * 9 + starter] == 0) continue;
        if (col_values.find(board[i * 9 + starter]) != col_values.end()) return 0;
        col_values.insert(board[i * 9 + starter]);
    }
    return true;
}

bool host_check_subsquare(int* board, int sub) {
    int col = (sub % 3) * 3, row = (sub / 3) * 3;
    std::set<int> square_values;
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            int starter = (row + i) * 9 + col + j;
            if (board[starter] == 0) continue;
            if (square_values.find(board[starter]) != square_values.end()) return 0;
            square_values.insert(board[starter]);
        }
    }
    return 1;
}

bool host_check_subsquare(int* board, int col, int row) {
    int sub = col / 3 + (row / 3) * 3;
    return host_check_subsquare(board, sub);
}

bool host_check_all(int* board) {
    bool format = true;
    for (int i = 0; i < 9; i++) {
        format = format && host_check_row(board, i);
    }
    for (int i = 0; i < 9; i++) {
        format = format && host_check_col(board, i);
    }
    for (int i = 0; i < 9; i++) {
        format = format && host_check_subsquare(board, i);
    }
    return format;
}

bool host_check_validity(int* board, int n, int col, int row) {
    for (int i = 0; i < 9; i++)
        if (board[row * 9 + i] == n || board[i * 9 + col] == n) return false;
    int sub_row_start = (row / 3) * 3;
    int sub_col_start = (col / 3) * 3;
    for (int i = sub_row_start; i < sub_row_start + 3; i++)
        for (int j = sub_col_start; j < sub_col_start + 3; j++)
            if (board[i * 9 + j] == n) return false;
    return true;
}

int host_solve_sudoku(int* board) {
    if (!host_check_all(board)) return -1; // Board is not valid
    for (int row = 0; row < 9; row++) {
        for (int col = 0; col < 9; col++) {
            int index = col + row * 9;
            if (!board[index]) {
                for (int i = 1; i <= 9; i++) {
                    if (host_check_validity(board, i, col, row)) {
                        board[index] = i;
                        if (host_solve_sudoku(board)) return 1;
                        else board[index] = 0;
                    }
                }
                return 0;
            }
        }
    }
    return 1;
}

void print_pointer(int* pointer, int size, int max_inline = -1) {
    if (max_inline <= 0) {
        for (int i = 0; i < size; i++) printf("\n%d", pointer[i]);
    }
    else {
        int flag = size / max_inline;
        int vals = size % max_inline == 0 ? flag : flag + 1;
        for (int i = 0; i < vals; i++) {
            printf("\n");
            for (int j = 0; j < max_inline; j++) {
                if (j + i * max_inline >= size) return;
                printf("%d ", pointer[j + i * max_inline]);
            }
        }
    }
}

__device__ bool device_check_col(int* board, int col) {
    bool col_values[9] = {false};
    for (int i = 0; i < 9; i++) {
        if (!board[i * 9 + col]) continue;
        if (col_values[board[i * 9 + col] - 1]) return false;
        else col_values[board[i * 9 + col] - 1] = true;
    }
    return true;
}

__device__ bool device_check_row(int* board, int row) {
    bool row_values[9] = { false };
    for (int i = 0; i < 9; i++) {
        if (!board[row * 9 + i]) continue;
        if (row_values[board[row * 9 + i] - 1]) return false;
        else row_values[board[row * 9 + i] - 1] = true;
    }
    return true;
}

__device__ bool device_check_subsquare(int* board, int square) {
    bool square_values[9] = { false };
    int row = (square / 3) * 3, col = (square % 3) * 3;
    int start_cell = row * 9 + col;
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            int index = start_cell + i * 9 + j;
            if (!board[index]) continue;
            if (square_values[board[index] - 1]) return false;
            else square_values[board[index] - 1] = true;
        }
    }
    return true;
}

__global__ void device_validity_all(int* board, bool* status) {
    int gid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    int oid = gid; //Operation id
    if (gid < 9) {
        status&& device_check_col(board, oid);
    }
    else if (gid < 18) {
        //TODO device check row
        oid -= 9;
        status&& device_check_row(board, oid);
    }
    else if (gid < 27) {
        //TODO device check subsquare
        oid -= 18;
        status&& device_check_subsquare(board, oid);
    }
}

int main()
{
    int* host_pointer;
    int* device_pointer;
    int* sudoku_board;
    dim3 blockSize(3, 3, 3);
    dim3 gridSize(1, 1, 1);
    // 81 because sudoku boards always have 81 values
    host_pointer = (int*)malloc(sizeof(int) * 81);
    hipMalloc((void**)&device_pointer, sizeof(int) * 81);
    sudoku_board = (int*)malloc(sizeof(int) * 81);
    for (int i = 0; i < 81; i++) {
        int row = i / 9;
        int col = i % 9;
        host_pointer[i] = board[row][col];
        sudoku_board[i] = board[row][col];
    }
    hipMemcpy(device_pointer, host_pointer, sizeof(int) * 81, hipMemcpyHostToDevice);
    bool status = true;
    device_validity_all << <gridSize, blockSize >> > (device_pointer, &status);
    hipDeviceSynchronize();
    if (!status) printf("\nThe format of the is not valid");
    else printf("\nFormat is valid");
    /*int board_status = host_solve_sudoku(host_pointer);
    if (board_status == -1) printf("\nThe format of the is not valid");
    else if (board_status == 0) printf("\nNo valid solution was found");
    else printf("\nA solution was found");*/
    //printf("\nThe format of the board is: %d", host_check_all(host_pointer));
    print_pointer(host_pointer, 81, 9);
    free(host_pointer);
    free(sudoku_board);
    hipFree(device_pointer);
}
